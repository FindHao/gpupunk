#include "hip/hip_runtime.h"
#include "gpu-patch.h"
#include "gpu-queue.h"
#include "utils.h"

#include <sanitizer_patching.h>

/*
 * Monitor each shared and global memory access.
 */
static __device__ __forceinline__
    SanitizerPatchResult
    memory_access_callback(
        void *user_data,
        uint64_t pc,
        void *address,
        uint32_t size,
        uint32_t flags,
        const void *new_value)
{
    gpu_patch_buffer_t *buffer = (gpu_patch_buffer_t *)user_data;

    if (!sample_callback(buffer->block_sampling_frequency, buffer->block_sampling_offset))
    {
        return SANITIZER_PATCH_SUCCESS;
    }

    // 1. Init values
    uint32_t active_mask = __activemask();
    uint32_t laneid = get_laneid();
    uint32_t first_laneid = __ffs(active_mask) - 1;

    // 2. Read memory values
    uint8_t buf[GPU_PATCH_MAX_ACCESS_SIZE];
    if (new_value == NULL)
    {
        // Read operation, old value can be on local memory, shared memory, or global memory
        if (flags & GPU_PATCH_SHARED)
        {
            // hidden type conversion, uintptr_t -> uint32
          read_shared_memory(size, (uintptr_t)address, buf);
        }
        else if (flags & GPU_PATCH_LOCAL)
        {
          read_local_memory(size, (uintptr_t)address, buf);
        }
        else if (flags != SANITIZER_MEMORY_DEVICE_FLAG_FORCE_INT)
        {
            read_global_memory(size, (uint64_t)address, buf);
        }
    }
    else
    {
        // Write operation, new value is on global memory
        read_global_memory(size, (uint64_t)new_value, buf);
    }

    gpu_patch_record_t *record = NULL;
    if (laneid == first_laneid)
    {
        // 3. Get a record
        gpu_patch_record_t *records = (gpu_patch_record_t *)buffer->records;
        record = records + gpu_queue_get(buffer, (buffer->flags & GPU_PATCH_ANALYSIS) != 0);

        // 4. Assign basic values
        record->flags = flags;
        record->active = active_mask;
        record->pc = pc;
        record->size = size;
        record->flat_thread_id = get_flat_thread_id();
        record->flat_block_id = get_flat_block_id();
    }

    __syncwarp(active_mask);

    uint64_t r = (uint64_t)record;
    record = (gpu_patch_record_t *)shfl(r, first_laneid, active_mask);

    if (record != NULL)
    {
        record->address[laneid] = (uint64_t)address;
        for (uint32_t i = 0; i < size; ++i)
        {
            record->value[laneid][i] = buf[i];
        }
    }

    __syncwarp(active_mask);

    if (laneid == first_laneid)
    {
        // 5. Push a record
        gpu_queue_push(buffer);
    }

    return SANITIZER_PATCH_SUCCESS;
}

extern "C" __device__ __noinline__
    SanitizerPatchResult
    sanitizer_global_memory_access_callback(
        void *user_data,
        uint64_t pc,
        void *address,
        uint32_t size,
        uint32_t flags,
        const void *new_value)
{
    return memory_access_callback(user_data, pc, address, size, flags, new_value);
}

extern "C" __device__ __noinline__
    SanitizerPatchResult
    sanitizer_shared_memory_access_callback(
        void *user_data,
        uint64_t pc,
        void *address,
        uint32_t size,
        uint32_t flags,
        const void *new_value)
{
    return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_SHARED, new_value);
}

extern "C" __device__ __noinline__
    SanitizerPatchResult
    sanitizer_local_memory_access_callback(
        void *user_data,
        uint64_t pc,
        void *address,
        uint32_t size,
        uint32_t flags,
        const void *new_value)
{
    return memory_access_callback(user_data, pc, address, size, flags | GPU_PATCH_LOCAL, new_value);
}

/*
 * Lock the corresponding hash entry for a block
 */
extern "C" __device__ __noinline__
    SanitizerPatchResult
    sanitizer_block_exit_callback(
        void *user_data,
        uint64_t pc)
{
    gpu_patch_buffer_t *buffer = (gpu_patch_buffer_t *)user_data;

    if (!sample_callback(buffer->block_sampling_frequency, buffer->block_sampling_offset))
    {
        return SANITIZER_PATCH_SUCCESS;
    }

    uint32_t active_mask = __activemask();
    uint32_t laneid = get_laneid();
    uint32_t first_laneid = __ffs(active_mask) - 1;
    int32_t pop_count = __popc(active_mask);

    if (laneid == first_laneid)
    {
        gpu_patch_record_t *records = (gpu_patch_record_t *)buffer->records;
        gpu_patch_record_t *record = records + gpu_queue_get(buffer, (buffer->flags & GPU_PATCH_ANALYSIS) != 0);

        record->pc = pc;
        record->flags = GPU_PATCH_BLOCK_EXIT_FLAG;
        record->flat_block_id = get_flat_block_id();
        record->flat_thread_id = get_flat_thread_id();
        record->active = active_mask;

        gpu_queue_push(buffer);

        // Finish a bunch of threads
        atomicAdd(&(buffer->num_threads), -pop_count);
    }

    return SANITIZER_PATCH_SUCCESS;
}

/*
 * Sample the corresponding blocks
 */
extern "C" __device__ __noinline__
    SanitizerPatchResult
    sanitizer_block_enter_callback(
        void *user_data,
        uint64_t pc)
{
    gpu_patch_buffer_t *buffer = (gpu_patch_buffer_t *)user_data;

    if (!sample_callback(buffer->block_sampling_frequency, buffer->block_sampling_offset))
    {
        return SANITIZER_PATCH_SUCCESS;
    }

    uint32_t active_mask = __activemask();
    uint32_t laneid = get_laneid();
    uint32_t first_laneid = __ffs(active_mask) - 1;

    if (laneid == first_laneid)
    {
        // Mark block begin
        gpu_patch_record_t *records = (gpu_patch_record_t *)buffer->records;
        gpu_patch_record_t *record = records + gpu_queue_get(buffer, (buffer->flags & GPU_PATCH_ANALYSIS) != 0);

        record->pc = pc;
        record->flags = GPU_PATCH_BLOCK_ENTER_FLAG;
        record->flat_block_id = get_flat_block_id();
        record->flat_thread_id = get_flat_thread_id();
        record->active = active_mask;

        gpu_queue_push(buffer);
    }

    return SANITIZER_PATCH_SUCCESS;
}
