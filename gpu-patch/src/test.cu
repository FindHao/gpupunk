#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "gpu-patch.h"
#include "utils.h"

#define GPU_ANALYSIS_DEBUG 1

#if GPU_ANALYSIS_DEBUG
#define PRINT(...)                         \
  if (threadIdx.x == 0 && blockIdx.x == 0) \
  {                                        \
    printf(__VA_ARGS__);                   \
  }
#define PRINT_ALL(...) \
  printf(__VA_ARGS__)
#define PRINT_RECORDS(buffer)                                                                                               \
  __syncthreads();                                                                                                          \
  if (threadIdx.x == 0)                                                                                                     \
  {                                                                                                                         \
    gpu_patch_analysis_address_t *records = (gpu_patch_analysis_address_t *)buffer->records;                                \
    for (uint32_t i = 0; i < buffer->head_index; ++i)                                                                       \
    {                                                                                                                       \
      printf("gpu analysis-> merged <%p, %p> (%p)\n", records[i].start, records[i].end, records[i].end - records[i].start); \
    }                                                                                                                       \
  }                                                                                                                         \
  __syncthreads();
#else
#define PRINT(...)
#define PRINT_ALL(...)
#define PRINT_RECORDS(buffer)
#endif

#define MAX_U64 (0xFFFFFFFFFFFFFFFF)
#define MAX_U32 (0xFFFFFFFF)

#define SANITIZER_FN_NAME(f) f

#define CHECK_CALL(fn, args)                         \
  {                                                  \
    hipError_t status = SANITIZER_FN_NAME(fn) args; \
    if (status != hipSuccess)                       \
    {                                                \
      fprintf(stderr, "error code %s\n",             \
              hipGetErrorString(status));           \
      exit(EXIT_FAILURE);                            \
    }                                                \
  }

/**
 * Each gpu_patch_buffer_t has a pointer to its records, and each records has 32 addresses. This function will unfold this structure into gpu_patch_buffer_t has new records while each record only has one address and its count.
 * @param buffer: the original buffer with a bunch of records
 * @param tmp_buffer: the buffer with unfolded records
 */
static __device__ void unfold_records(gpu_patch_buffer_t *patch_buffer, gpu_patch_buffer_t *tmp_buffer)
{
  auto warp_index = blockDim.x / GPU_PATCH_WARP_SIZE * blockIdx.x + threadIdx.x / GPU_PATCH_WARP_SIZE;
  // by default it is 4
  auto num_warps = blockDim.x / GPU_PATCH_WARP_SIZE;
  auto laneid = get_laneid();
  gpu_patch_record_address_t *records = (gpu_patch_record_address_t *)patch_buffer->records;
  gpu_patch_addr_sort_t *addr_hist = (gpu_patch_addr_sort_t *)tmp_buffer->records;
  PRINT("gpu analysis->full: %u, analysis: %u, head_index: %u, tail_index: %u, size: %u, num_threads: %u",
        patch_buffer->full, patch_buffer->analysis, patch_buffer->head_index, patch_buffer->tail_index,
        patch_buffer->size, patch_buffer->num_threads)
  // each warp will take care with one record (32 addresses) in each iteration
  for (auto iter = warp_index; iter < patch_buffer->head_index; iter += num_warps)
  {
    gpu_patch_record_address_t *record = records + iter;
    uint64_t address = record->address[laneid];
    // if the thread is not active, set the address to 0
    if (((0x1u << laneid) & record->active) == 0)
    {
      address = 0;
    }

    addr_hist[iter * GPU_PATCH_WARP_SIZE + laneid] = address;
  }
  tmp_buffer->head_index = patch_buffer->head_index * GPU_PATCH_WARP_SIZE;
}
template <int THREADS>
static __device__ void block_radix_sort(gpu_patch_buffer_t *tmp_buffer, gpu_patch_buffer_t *hist_buffer) {
  int num_of_records = tmp_buffer->head_index;
  // DEFAULT_GPU_PATCH_RECORD_NUM is 1280*1024 by default. each record includes 32 addresses with uint64_t type. so the total size is 1280*1024*32*8 = 335544320 bytes. Since we have 1024 threads for our analysis kernel, each thread need 335544320/1024/1024 = 320KB memory. The max local memory per thread is 512KB, so we are good for default configuration.
  int items_per_thread = num_of_records / THREADS;
  // Specialize BlockRadixSort type for our thread block
  typedef hipcub::BlockRadixSort<uint64_t, THREADS, items_per_thread, uint64_t> BlockRadixSortT;
  // __shared__ typename BlockRadixSort::TempStorage temp_storage;
  uint64_t *keys_in = (uint64_t *)tmp_buffer->records;
}

extern "C" __launch_bounds__(GPU_PATCH_ANALYSIS_THREADS, 1)
    __global__
    void gpu_analysis_hist(
        gpu_patch_buffer_t *buffer,
        gpu_patch_buffer_t *tmp_buffer
        // gpu_patch_buffer_t *hist_buffer
    )
{
  // // Continue processing until CPU notifies analysis is done
  // while (true) {
  //   // Wait until GPU notifies buffer is full. i.e., analysis can begin process.
  //   // Block sampling is not allowed
  //   while (buffer->analysis == 0 && atomic_load(&buffer->num_threads) != 0)
  //     ;
  //   if (atomic_load(&buffer->num_threads) == 0) {
  //     // buffer->analysis must be 0
  //     break;
  //   }

  // }
  unfold_records(buffer, tmp_buffer);
}

int main(int argc, char **argv)
{
  std::cout << "Hello, world!" << std::endl;
  int num_records = 2000;

  // tmp_buffer is used to store the unfolded records
  gpu_patch_buffer_t *tmp_buffer;
  CHECK_CALL(hipMalloc, ((void **)&tmp_buffer, sizeof(gpu_patch_buffer_t)));
  void *tmp_buffer_records_g = NULL;
  CHECK_CALL(hipMalloc, ((void **)&tmp_buffer_records_g,
                          sizeof(gpu_patch_addr_sort_t) * num_records * GPU_PATCH_WARP_SIZE));
  // we need to update the records pointer in tmp_buffer by this way. because we can't directly update the records pointer in tmp_buffer on CPU side.
  gpu_patch_buffer_t *tmp_buffer_h;
  tmp_buffer_h = (gpu_patch_buffer_t *)malloc(sizeof(gpu_patch_buffer_t));
  tmp_buffer_h->records = tmp_buffer_records_g;

  CHECK_CALL(hipMemcpy, (tmp_buffer, tmp_buffer_h, sizeof(gpu_patch_buffer_t), hipMemcpyHostToDevice));

  // gpu_buffer stores the original trace
  gpu_patch_buffer_t *gpu_buffer;
  CHECK_CALL(hipMalloc, ((void **)&gpu_buffer, sizeof(gpu_patch_buffer_t)));
  void *gpu_buffer_records;
  CHECK_CALL(hipMalloc, ((void **)&gpu_buffer_records,
                          sizeof(gpu_patch_record_address_t) * num_records));

  gpu_patch_buffer_t *gpu_buffer_h;
  gpu_buffer_h = (gpu_patch_buffer_t *)malloc(sizeof(gpu_patch_buffer_t));
  gpu_patch_record_address_t *gpu_buffer_records_h;
  gpu_buffer_records_h = (gpu_patch_record_address_t *)malloc(sizeof(gpu_patch_record_address_t) * num_records);
  gpu_buffer_h->records = gpu_buffer_records;
  gpu_buffer_h->head_index = num_records;
  for (int i = 0; i < num_records; i++)
  {
    for (int j = 0; j < GPU_PATCH_WARP_SIZE; j++)
    {
      gpu_buffer_records_h[i].address[j] = i % 100;
      gpu_buffer_records_h[i].size = 1;
    }
    gpu_buffer_records_h[i].active = 0xffffffff;
  }
  CHECK_CALL(hipMemcpy, (gpu_buffer, gpu_buffer_h, sizeof(gpu_patch_buffer_t), hipMemcpyHostToDevice));
  CHECK_CALL(hipMemcpy, (gpu_buffer_records, gpu_buffer_records_h, sizeof(gpu_patch_record_address_t) * num_records, hipMemcpyHostToDevice));
  gpu_analysis_hist<<<1, GPU_PATCH_ANALYSIS_THREADS>>>(gpu_buffer, tmp_buffer);
  gpu_patch_addr_sort_t *tmp_buffer_records_h = (gpu_patch_addr_sort_t *)malloc(sizeof(gpu_patch_addr_sort_t) * num_records * GPU_PATCH_WARP_SIZE);
  CHECK_CALL(hipMemcpy, (tmp_buffer_records_h, tmp_buffer_records_g, sizeof(gpu_patch_addr_sort_t) * num_records * GPU_PATCH_WARP_SIZE, hipMemcpyDeviceToHost));
  CHECK_CALL(hipDeviceSynchronize, ());
  for (int i = 0; i < num_records; i++)
  {
    for (int j = 0; j < GPU_PATCH_WARP_SIZE; j++)
    {
      std::cout << tmp_buffer_records_h[i * GPU_PATCH_WARP_SIZE + j] << " ";
    }
    std::cout << std::endl;
  }

  CHECK_CALL(hipFree, (gpu_buffer));
  CHECK_CALL(hipFree, (gpu_buffer_records));
  CHECK_CALL(hipFree, (tmp_buffer));
  CHECK_CALL(hipFree, (tmp_buffer_records_g));
  free(tmp_buffer_h);
  free(gpu_buffer_h);
  free(gpu_buffer_records_h);

  return 0;
}