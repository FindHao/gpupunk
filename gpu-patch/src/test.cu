#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "gpu-patch.h"
#include "utils.h"
#include <hipcub/hipcub.hpp>

#define GPU_ANALYSIS_DEBUG 1

#if GPU_ANALYSIS_DEBUG
#define PRINT(...)                         \
  if (threadIdx.x == 0 && blockIdx.x == 0) \
  {                                        \
    printf(__VA_ARGS__);                   \
  }
#define PRINT_ALL(...) \
  printf(__VA_ARGS__)
#define PRINT_RECORDS(buffer)                                                                                               \
  __syncthreads();                                                                                                          \
  if (threadIdx.x == 0)                                                                                                     \
  {                                                                                                                         \
    gpu_patch_analysis_address_t *records = (gpu_patch_analysis_address_t *)buffer->records;                                \
    for (uint32_t i = 0; i < buffer->head_index; ++i)                                                                       \
    {                                                                                                                       \
      printf("gpu analysis-> merged <%p, %p> (%p)\n", records[i].start, records[i].end, records[i].end - records[i].start); \
    }                                                                                                                       \
  }                                                                                                                         \
  __syncthreads();
#else
#define PRINT(...)
#define PRINT_ALL(...)
#define PRINT_RECORDS(buffer)
#endif

#define MAX_U64 (0xFFFFFFFFFFFFFFFF)
#define MAX_U32 (0xFFFFFFFF)

#define SANITIZER_FN_NAME(f) f

#define CHECK_CALL(fn, args)                         \
  {                                                  \
    hipError_t status = SANITIZER_FN_NAME(fn) args; \
    if (status != hipSuccess)                       \
    {                                                \
      fprintf(stderr, "error in %s\n", #fn);         \
      fprintf(stderr, "error code %s\n",             \
              hipGetErrorString(status));           \
      exit(EXIT_FAILURE);                            \
    }                                                \
  }

/**
 * Each gpu_patch_buffer_t has a pointer to its records, and each records has 32 addresses. This function will unfold this structure into gpu_patch_buffer_t has new records while each record only has one address and its count. Besides the unfolding, this function will also do intra-warp counting.
 * @param buffer: the original buffer with a bunch of records
 * @param unfolded_buffer: the buffer with unfolded and intra-warp-processed records.
 */
static __device__ void unfold_records(gpu_patch_buffer_t *patch_buffer, gpu_patch_buffer_t *unfolded_buffer)
{
  auto warp_id = blockDim.x / GPU_PATCH_WARP_SIZE * blockIdx.x + threadIdx.x / GPU_PATCH_WARP_SIZE;
  // by default it is 4
  auto num_warps = blockDim.x / GPU_PATCH_WARP_SIZE;
  auto laneid = get_laneid();
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  gpu_patch_record_address_t *records = (gpu_patch_record_address_t *)patch_buffer->records;
  gpu_patch_addr_hist_t *addr_hist = (gpu_patch_addr_hist_t *)unfolded_buffer->records;
  PRINT("gpu analysis->full: %u, analysis: %u, head_index: %u, tail_index: %u, size: %u, num_threads: %u",
        patch_buffer->full, patch_buffer->analysis, patch_buffer->head_index, patch_buffer->tail_index,
        patch_buffer->size, patch_buffer->num_threads)
  int addr_hist_index = 0;
  auto iter = warp_id;
  int round_head_index = (patch_buffer->head_index + num_warps - 1) / num_warps * num_warps;
  // every record has 32 addresses with mask. but the number of record may not be `num_warps` aligned.
  // e.g., there are 3 records. we need to let the last warp be inactive.
  // each warp will take care with one record (32 addresses) in each iteration
  for (; iter < round_head_index; iter += num_warps)
  {
    if (iter >= patch_buffer->head_index)
    {
      continue;
    }
    gpu_patch_record_address_t *record = records + iter;
    uint64_t address = record->address[laneid];
    // if the thread is not active, set the address to 0
    if (((0x1u << laneid) & record->active) == 0)
    {
      address = 0;
    }
    // sort addresses of a record inside a warp
    address = warp_sort(address, laneid);

    uint64_t value = address;
    uint64_t prev_value = __shfl_up_sync(0xffffffff, value, 1);
    bool is_unique = (laneid == 0) || (value != prev_value);

    unsigned int predicate = __ballot_sync(0xffffffff, !is_unique);
    uint32_t mask = 0xFFFFFFFF << laneid;
    uint32_t predicate2 = predicate & mask;
    uint32_t predicate2_reverse = __brev(predicate2);
    int leading_zeros_origin = __clz(predicate2_reverse);
    // Find the position of the most significant bit (MSB) set to 1
    int msb_position = 31 - __clz(predicate2_reverse);
    // Create a mask with all bits set to 1 up to and including the MSB
    mask = (0xFFFFFFFF >> (31 - msb_position));
    // Invert the input number (flip all the bits)
    unsigned int inverted_x = ~predicate2_reverse;
    // Count the number of leading zeros in the inverted number masked with the mask
    int leading_zeros = __clz(inverted_x & mask);
    int leading_ones = (predicate & (1 << (laneid + 1))) ? leading_zeros - leading_zeros_origin : 0;
    int count = leading_ones + 1;
    // how many unique addresses in this warp
    int unique_mark = __ballot_sync(0xffffffff, is_unique);
    __shared__ int unique_count_shared[GPU_PATH_ANALYSIS_NUM_WARPS];
    __shared__ int unique_count_shared_accumulate[GPU_PATH_ANALYSIS_NUM_WARPS];
    if (laneid == 0)
    {
      unique_count_shared[warp_id] = __popc(unique_mark);
      // unique_count_shared_accumulate[warp_id] = __popc(unique_mark);
      if (warp_id == 0)
      {
        int next_start = 0;
        for (int i = 0; i < GPU_PATH_ANALYSIS_NUM_WARPS; i++)
        {
          unique_count_shared_accumulate[i] = next_start;
          next_start += unique_count_shared[i];
        }
      }
    }
    __shared__ uint64_t addr_hist_addr[GPU_PATCH_WARP_SIZE * GPU_PATH_ANALYSIS_NUM_WARPS];
    __shared__ int addr_hist_count[GPU_PATCH_WARP_SIZE * GPU_PATH_ANALYSIS_NUM_WARPS];
    __syncthreads();
    if (is_unique)
    {
      int output_idx = __popc(unique_mark & ((1 << laneid) - 1)) + unique_count_shared_accumulate[warp_id];
      addr_hist_addr[output_idx] = value;
      addr_hist_count[output_idx] = count;
    }
    __syncthreads();
    if (idx == 0)
    {
      int all_unique_count = unique_count_shared_accumulate[GPU_PATH_ANALYSIS_NUM_WARPS - 1] + unique_count_shared[GPU_PATH_ANALYSIS_NUM_WARPS - 1];
      for (int i = 0; i < all_unique_count; i++)
      {
        addr_hist[addr_hist_index + i].address = addr_hist_addr[i];
        addr_hist[addr_hist_index + i].count = addr_hist_count[i];
      }
      addr_hist_index += all_unique_count;
    }
  }
  // unfolded_buffer->head_index = patch_buffer->head_index * GPU_PATCH_WARP_SIZE;
  unfolded_buffer->head_index = addr_hist_index;
}

/**
 * @brief This function only sorts THREADS * ITEMS_PER_THREAD items in unfolded_buffer->records
 * @Yueming TODO: add the histogram part
 */
template <int THREADS, int ITEMS_PER_THREAD>
static __device__ void block_radix_sort_tile(
    uint64_t *d_in,
    uint64_t *d_out)
{
  typedef hipcub::BlockRadixSort<uint64_t, THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
  __shared__ typename BlockRadixSortT::TempStorage temp_storage;
  uint64_t keys[ITEMS_PER_THREAD];
  for (int i = 0; i < ITEMS_PER_THREAD; ++i)
  {
    keys[i] = d_in[threadIdx.x * ITEMS_PER_THREAD + i];
  }
  BlockRadixSortT(temp_storage).Sort(keys);
  for (int i = 0; i < ITEMS_PER_THREAD; ++i)
  {
    d_out[threadIdx.x * ITEMS_PER_THREAD + i] = keys[i];
  }
  // maybe we can use the similar code in unfold_records to process all 4 warps in a block.
}

template <int THREADS, int ITEMS_PER_THREAD>
static __device__ void block_radix_sort(
    gpu_patch_buffer_t *unfolded_buffer,
    gpu_patch_buffer_t *hist_buffer)
{
}

extern "C" __launch_bounds__(GPU_PATCH_ANALYSIS_THREADS, 1)
    __global__
    void gpu_analysis_hist(
        gpu_patch_buffer_t *buffer,
        gpu_patch_buffer_t *unfolded_buffer,
        gpu_patch_addr_hist_t *unfolded_buffer_records_g_sorted
        // gpu_patch_buffer_t *hist_buffer
    )
{
  // // Continue processing until CPU notifies analysis is done
  // while (true) {
  //   // Wait until GPU notifies buffer is full. i.e., analysis can begin process.
  //   // Block sampling is not allowed
  //   while (buffer->analysis == 0 && atomic_load(&buffer->num_threads) != 0)
  //     ;
  //   if (atomic_load(&buffer->num_threads) == 0) {
  //     // buffer->analysis must be 0
  //     break;
  //   }

  // }
  unfold_records(buffer, unfolded_buffer);
  // @Yueming TODO: use a for loop to split the unfolded_buffer into multiple tiles, and use block_radix_sort_tile to process each tile. Add another outside for loop to process at least twice to compress more. Finally, the unfolded_buffer_records_g_sorted will have compressed histogram.
  // uint32_t tile_size = THREADS * GPU_PATCH_ANALYSIS_THREADS;
  // block_radix_sort<GPU_PATCH_ANALYSIS_THREADS, GPU_PATCH_ANALYSIS_ITEMS>(unfolded_buffer, hist_buffer);
}

int main(int argc, char **argv)
{
  std::cout << "Hello, world!" << std::endl;
  int num_records = 3;

  // unfolded_buffer is used to store the unfolded records
  gpu_patch_buffer_t *unfolded_buffer;
  CHECK_CALL(hipMalloc, ((void **)&unfolded_buffer, sizeof(gpu_patch_buffer_t)));
  // unfolded_buffer_records_g is used to store the unfolded records
  void *unfolded_buffer_records_g = NULL;
  CHECK_CALL(hipMalloc, ((void **)&unfolded_buffer_records_g,
                          sizeof(gpu_patch_addr_hist_t) * num_records * GPU_PATCH_WARP_SIZE));
  // unfolded_buffer_records_g_sorted is used to store the sorted unfolded records
  void *unfolded_buffer_records_g_sorted = NULL;
  CHECK_CALL(hipMalloc, ((void **)&unfolded_buffer_records_g_sorted,
                          sizeof(gpu_patch_addr_hist_t) * num_records * GPU_PATCH_WARP_SIZE));
  // we need to update the records pointer in unfolded_buffer by this way. because we can't directly update the records pointer in unfolded_buffer on CPU side.
  gpu_patch_buffer_t *unfolded_buffer_h;
  unfolded_buffer_h = (gpu_patch_buffer_t *)malloc(sizeof(gpu_patch_buffer_t));
  unfolded_buffer_h->records = unfolded_buffer_records_g;

  CHECK_CALL(hipMemcpy, (unfolded_buffer, unfolded_buffer_h, sizeof(gpu_patch_buffer_t), hipMemcpyHostToDevice));

  // gpu_buffer stores the original trace
  gpu_patch_buffer_t *gpu_buffer;
  CHECK_CALL(hipMalloc, ((void **)&gpu_buffer, sizeof(gpu_patch_buffer_t)));
  void *gpu_buffer_records;
  CHECK_CALL(hipMalloc, ((void **)&gpu_buffer_records,
                          sizeof(gpu_patch_record_address_t) * num_records));

  gpu_patch_buffer_t *gpu_buffer_h;
  gpu_buffer_h = (gpu_patch_buffer_t *)malloc(sizeof(gpu_patch_buffer_t));
  gpu_patch_record_address_t *gpu_buffer_records_h;
  gpu_buffer_records_h = (gpu_patch_record_address_t *)malloc(sizeof(gpu_patch_record_address_t) * num_records);
  gpu_buffer_h->records = gpu_buffer_records;
  gpu_buffer_h->head_index = num_records;
  for (int i = 0; i < num_records; i++)
  {
    for (int j = 0; j < GPU_PATCH_WARP_SIZE; j++)
    {
      // gpu_buffer_records_h[i].address[j] = j % 10;
      gpu_buffer_records_h[i].address[j] = 1;
      gpu_buffer_records_h[i].size = 1;
    }
    gpu_buffer_records_h[i].active = 0xffffffff;
  }
  CHECK_CALL(hipMemcpy, (gpu_buffer, gpu_buffer_h, sizeof(gpu_patch_buffer_t), hipMemcpyHostToDevice));
  CHECK_CALL(hipMemcpy, (gpu_buffer_records, gpu_buffer_records_h, sizeof(gpu_patch_record_address_t) * num_records, hipMemcpyHostToDevice));
  gpu_analysis_hist<<<1, GPU_PATCH_ANALYSIS_THREADS>>>(gpu_buffer, unfolded_buffer, (gpu_patch_addr_hist_t *)unfolded_buffer_records_g_sorted);

  gpu_patch_addr_hist_t *unfolded_buffer_records_h = (gpu_patch_addr_hist_t *)malloc(sizeof(gpu_patch_addr_hist_t) * num_records * GPU_PATCH_WARP_SIZE);
  // copy the unfolded records from GPU to CPU
  CHECK_CALL(hipMemcpy, (unfolded_buffer_records_h, unfolded_buffer_records_g, sizeof(gpu_patch_addr_hist_t) * num_records * GPU_PATCH_WARP_SIZE, hipMemcpyDeviceToHost));
  // copy the head_index back to CPU
  CHECK_CALL(hipMemcpy, (unfolded_buffer_h, unfolded_buffer, sizeof(gpu_patch_buffer_t), hipMemcpyDeviceToHost));
  CHECK_CALL(hipDeviceSynchronize, ());
  std::cout << std::endl
            << "unfolded records: "
            << "head_index:" << unfolded_buffer_h->head_index << std::endl;
  for (int i = 0; i < num_records; i++)
  {
    for (int j = 0; j < GPU_PATCH_WARP_SIZE; j++)
    {
      std::cout << unfolded_buffer_records_h[i * GPU_PATCH_WARP_SIZE + j].address << ":" << unfolded_buffer_records_h[i * GPU_PATCH_WARP_SIZE + j].count << "  ";
    }
    std::cout << std::endl;
  }

  CHECK_CALL(hipFree, (gpu_buffer));
  CHECK_CALL(hipFree, (gpu_buffer_records));
  CHECK_CALL(hipFree, (unfolded_buffer));
  CHECK_CALL(hipFree, (unfolded_buffer_records_g));
  free(unfolded_buffer_h);
  free(gpu_buffer_h);
  free(gpu_buffer_records_h);

  return 0;
}